#include "hip/hip_runtime.h"
/*
 *  gpu_kernels.cu -- GPU kernels
 *
 *  Copyright (C) 2014, Computing Systems Laboratory (CSLab)
 *  Copyright (C) 2014, Vasileios Karakasis
 */ 

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "error.h"
#include "gpu_util.h"
#include "graph.h"
#include "timer.h"
#include "alloc.h"

#define GPU_KERNEL_NAME(name)   do_apsp_gpu ## name
#define BLOCK_SIZE 32
#define GRID_SIZE n

#define TILE(a,b,i,j) dist[(a*GPU_TILE_DIM+i)*n+(b*GPU_TILE_DIM+j)]
weight_t *copy_graph_to_gpu(const graph_t *graph)
{
    size_t dist_size = graph->nr_vertices*graph->nr_vertices;
    weight_t *dist_gpu = (weight_t *) gpu_alloc(dist_size*sizeof(*dist_gpu));
    if (!dist_gpu)
        error(0, "gpu_alloc() failed: %s", gpu_get_last_errmsg());

    if (copy_to_gpu(graph->weights[0], dist_gpu,
                    dist_size*sizeof(*dist_gpu)) < 0)
        error(0, "copy_to_gpu() failed: %s", gpu_get_last_errmsg());

    return dist_gpu;
}

graph_t *copy_graph_from_gpu(const weight_t *dist_gpu, graph_t *graph)
{
    size_t dist_size = graph->nr_vertices*graph->nr_vertices;

    if (copy_from_gpu(graph->weights[0], dist_gpu,
                      dist_size*sizeof(*dist_gpu)) < 0)
        error(0, "copy_from_gpu() failed: %s", gpu_get_last_errmsg());

    return graph;
}



/***************************************************************************************/
/*****					KERNEL: Naive				   *****/
/***************************************************************************************/
/*
 * The naive GPU kernel
 */

__global__ void GPU_KERNEL_NAME(_naive)(weight_t *dist, int n, int k){
    uint tidx = blockIdx.x * blockDim.x + threadIdx.x;
    uint tidy = blockIdx.y * blockDim.y + threadIdx.y;
    
    dist[tidy*n + tidx] = MIN(dist[tidy*n + tidx], dist[tidy*n + k] + dist[k*n + tidx]);
}


/***************************************************************************************/
/*****			KERNEL: Tiled - Global Memory				   *****/
/***************************************************************************************/
/*
 *  The tiled GPU kernel(s) using global memory
 */ 

__device__ void FWI_tiled(weight_t *dist, int a, int b, int c, int d, int e, int f, int n){
    uint k;
    for (k=0; k<GPU_TILE_DIM; k++){
	TILE(a, b, threadIdx.y, threadIdx.x) = MIN( TILE(a, b, threadIdx.y, threadIdx.x), TILE(c, d, threadIdx.y, k) + TILE(e, f, k, threadIdx.x) );
	__syncthreads();
    }
}

__global__ void GPU_KERNEL_NAME(_tiled_stage_1)(weight_t *dist, int n, int k)
{
    FWI_tiled(dist, k,k, k,k, k,k, n);
}

/* duo seires apo blocks. h prwth ypologizei thn k_row kai h deuterh thn k_line */
__global__ void GPU_KERNEL_NAME(_tiled_stage_2)(weight_t *dist, int n, int k)
{
    if (blockIdx.x == k)
	return;
    if (blockIdx.y == 0)
	FWI_tiled(dist, blockIdx.x,k, blockIdx.x,k, k,k, n);
    else
	FWI_tiled(dist, k,blockIdx.x, k,k, k,blockIdx.x, n);
}

__global__ void GPU_KERNEL_NAME(_tiled_stage_3)(weight_t *dist, int n,
                                                int k)
{
    if (blockIdx.x == k)
	return;
    if (blockIdx.y == k)
	return;
    
    FWI_tiled(dist, blockIdx.y,blockIdx.x, blockIdx.y,k, k,blockIdx.x, n);
}





/***************************************************************************************/
/*****			KERNEL: Tiled - Shared Memory				   *****/
/***************************************************************************************/
__device__ void FWI_shared_tiled(weight_t tile_1[][GPU_TILE_DIM], weight_t tile_2[][GPU_TILE_DIM], weight_t tile_3[][GPU_TILE_DIM]){
    uint k;
    for (k=0; k<GPU_TILE_DIM; k++){
	tile_1[threadIdx.y][threadIdx.x] = MIN(tile_1[threadIdx.y][threadIdx.x], tile_2[threadIdx.y][k] + tile_3[k][threadIdx.x]);
	__syncthreads();
    }
}

__device__ void fetch(int a, int b, weight_t tile[][GPU_TILE_DIM], weight_t *dist, int n){
    tile[threadIdx.y][threadIdx.x] = TILE(a,b, threadIdx.y, threadIdx.x);
}

__device__ void send(weight_t tile[][GPU_TILE_DIM], int a, int b, weight_t *dist, int n){
    TILE(a,b, threadIdx.y, threadIdx.x) = tile[threadIdx.y][threadIdx.x];
}

__global__ void GPU_KERNEL_NAME(_tiled_shmem_stage_1)(weight_t *dist, int n, int k){
    __shared__ weight_t tile_k[GPU_TILE_DIM][GPU_TILE_DIM];

    fetch(k,k, tile_k, dist, n);
    __syncthreads();
    
    FWI_shared_tiled(tile_k, tile_k, tile_k);
    send(tile_k, k,k, dist, n);
}

/* opws ston tiled-kernel, exoume 2 grammes apo nr_tiles blocks */
__global__ void GPU_KERNEL_NAME(_tiled_shmem_stage_2)(weight_t *dist, int n, int k){
    if (blockIdx.x == k)
	return;
    
    __shared__ weight_t tile_k[GPU_TILE_DIM][GPU_TILE_DIM];
    __shared__ weight_t my_tile[GPU_TILE_DIM][GPU_TILE_DIM];
    
    fetch (k,k, tile_k, dist, n);
    __syncthreads();
    
    if (blockIdx.y == 0){
	fetch(blockIdx.x, k, my_tile, dist, n);
	__syncthreads();
	
	FWI_shared_tiled(my_tile, my_tile, tile_k);
	send(my_tile, blockIdx.x, k, dist, n);
    }
    else {
	fetch(k, blockIdx.x, my_tile, dist, n);
	__syncthreads();
	
	FWI_shared_tiled(my_tile, tile_k, my_tile);
	send(my_tile, k, blockIdx.x, dist, n);
    }
}

__global__ void GPU_KERNEL_NAME(_tiled_shmem_stage_3)(weight_t *dist, int n, int k){
    if ((blockIdx.x == k) || (blockIdx.y == k))
	return;
    
    __shared__ weight_t tile_ij[GPU_TILE_DIM][GPU_TILE_DIM];
    __shared__ weight_t tile_ik[GPU_TILE_DIM][GPU_TILE_DIM];
    __shared__ weight_t tile_kj[GPU_TILE_DIM][GPU_TILE_DIM];
    
    fetch(blockIdx.y, blockIdx.x, tile_ij, dist, n);
    fetch(blockIdx.y, k, tile_ik, dist, n);
    fetch(k, blockIdx.x, tile_kj, dist, n);
    __syncthreads();
    FWI_shared_tiled(tile_ij, tile_ik, tile_kj);
    send(tile_ij, blockIdx.y, blockIdx.x, dist, n);
}

/*
 *  FILLME: Use different kernels for the different stages of the
 *  tiled FW computation
 *  
 *  Use GPU_TILE_DIM (see graph.h) as the tile dimension. You can
 *  adjust its value during compilation. See `make help' for more
 *  information.
 */ 


/***************************************************************************************/
/*****				CPU: Call Kernels				   *****/
/***************************************************************************************/

/***************************************************************************************/
/*****				Call: Naive Kernel				   *****/
/***************************************************************************************/

graph_t *MAKE_KERNEL_NAME(_gpu, _naive)(graph_t *graph)
{
    xtimer_t transfer_timer;
    timer_clear(&transfer_timer);
    timer_start(&transfer_timer);
    weight_t *dist_gpu = copy_graph_to_gpu(graph);
    timer_stop(&transfer_timer);

    /* FILLME: Set up and launch the kernel(s) */
    
    int k;
    int n = graph->nr_vertices;
    dim3 block(32, 32);
    dim3 grid(n/32, n/32);
    
    for (k=0; k<n; k++){
	GPU_KERNEL_NAME(_naive)<<<grid, block>>>(dist_gpu, n, k);
	hipDeviceSynchronize();
    }
    
    /*
     * Wait for last kernel to finish, so as to measure correctly the
     * transfer times Otherwise, copy from GPU will block
     */
//     hipDeviceSynchronize();

    /* Copy back results to host */
    timer_start(&transfer_timer);
    copy_graph_from_gpu(dist_gpu, graph);
    timer_stop(&transfer_timer);
    printf("Total transfer times: %lf s\n",
           timer_elapsed_time(&transfer_timer));
    return graph;
}


/***************************************************************************************/
/*****				Call: Tiled Kernel				   *****/
/***************************************************************************************/

graph_t *MAKE_KERNEL_NAME(_gpu, _tiled)(graph_t *graph)
{
    xtimer_t transfer_timer;
    timer_clear(&transfer_timer);
    timer_start(&transfer_timer);
    weight_t *dist_gpu = copy_graph_to_gpu(graph);
    timer_stop(&transfer_timer);

    uint N = graph->nr_vertices;
    uint nr_tiles = N / GPU_TILE_DIM;
    dim3 block(GPU_TILE_DIM, GPU_TILE_DIM);
    dim3 grid1(1);
    dim3 grid2(nr_tiles, 2);
    dim3 grid3(nr_tiles, nr_tiles);
    uint k;
    
    for (k=0; k<nr_tiles; k++){
	GPU_KERNEL_NAME(_tiled_stage_1)<<<grid1, block>>>(dist_gpu, N, k);
	hipDeviceSynchronize();
	
	GPU_KERNEL_NAME(_tiled_stage_2)<<<grid2, block>>>(dist_gpu, N, k);
	hipDeviceSynchronize();
	
	GPU_KERNEL_NAME(_tiled_stage_3)<<<grid3, block>>>(dist_gpu, N, k);
	hipDeviceSynchronize();
    }
    /*
     * Wait for last kernel to finish, so as to measure correctly the
     * transfer times Otherwise, copy from GPU will block
     */
//     hipDeviceSynchronize();

    /* Copy back results to host */
    timer_start(&transfer_timer);
    copy_graph_from_gpu(dist_gpu, graph);
    timer_stop(&transfer_timer);
    printf("Total transfer times: %lf s\n",
           timer_elapsed_time(&transfer_timer));
    return graph;
}

/***************************************************************************************/
/*****				Call: Tiled ShMem Kernel			   *****/
/***************************************************************************************/
graph_t *MAKE_KERNEL_NAME(_gpu, _tiled_shmem)(graph_t *graph)
{
    xtimer_t transfer_timer;
    timer_clear(&transfer_timer);
    timer_start(&transfer_timer);
    weight_t *dist_gpu = copy_graph_to_gpu(graph);
    timer_stop(&transfer_timer);

    uint N = graph->nr_vertices;
    uint nr_tiles = N / GPU_TILE_DIM;
    dim3 block(GPU_TILE_DIM, GPU_TILE_DIM);
    dim3 grid1(1);
    dim3 grid2(nr_tiles, 2);
    dim3 grid3(nr_tiles, nr_tiles);
    uint k;
    for (k=0; k<nr_tiles ;k++){
	
	GPU_KERNEL_NAME(_tiled_shmem_stage_1)<<<grid1, block>>>(dist_gpu, N, k);
	hipDeviceSynchronize();
	
	GPU_KERNEL_NAME(_tiled_shmem_stage_2)<<<grid2, block>>>(dist_gpu, N, k);
	hipDeviceSynchronize();
	
	GPU_KERNEL_NAME(_tiled_shmem_stage_3)<<<grid3, block>>>(dist_gpu, N, k);
	hipDeviceSynchronize();
    }
    /*
     * Wait for last kernel to finish, so as to measure correctly the
     * transfer times Otherwise, copy from GPU will block
     */
//     hipDeviceSynchronize();

    /* Copy back results to host */
    timer_start(&transfer_timer);
    copy_graph_from_gpu(dist_gpu, graph);
    timer_stop(&transfer_timer);
    printf("Total transfer times: %lf s\n",
           timer_elapsed_time(&transfer_timer));
    return graph;
}
